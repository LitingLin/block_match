#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

__global__ void
standardize_block_kernel(float *data, int blockSize)
{
	const int tid = threadIdx.x + blockDim.x * blockIdx.x;

	float *c_data = data + tid * blockSize;

	float mean = 0;
	for (int i = 0; i < blockSize; ++i)
	{
		mean += c_data[i];
	}
	mean /= blockSize;

	float sd = 0;
	for (int i = 0; i < blockSize; ++i)
	{
		float d = c_data[i] -= mean;
		sd += d*d;
	}

	sd /= blockSize;
	sd = sqrt(sd);

	for (int i = 0; i < blockSize; ++i)
	{
		c_data[i] /= sd;
	}
}

__global__ void
standardize_block_kernel(float *data, int blockSize, int n)
{
	const int tid = threadIdx.x + blockDim.x * blockIdx.x;

	if (tid >= n)
		return;

	float *c_data = data + tid * blockSize;

	float mean = 0;
	for (int i = 0; i < blockSize; ++i)
	{
		mean += c_data[i];
	}
	mean /= blockSize;

	float sd = 0;
	for (int i = 0; i < blockSize; ++i)
	{
		float d = c_data[i] -= mean;
		sd += d*d;
	}

	sd /= blockSize;
	sd = sqrtf(sd);

	for (int i = 0; i < blockSize; ++i)
	{
		c_data[i] /= sd;
	}
}

__global__ void
vector_multiply_add(float *block_A, float *block_B, int blockSize, float *result)
{
	const int tid = threadIdx.x + blockDim.x * blockIdx.x;

	const float *c_block_A = block_A + blockIdx.x * blockSize;
	const float *c_block_B = block_B + threadIdx.x * blockSize;

	float temp = 0;
	for (int i = 0; i < blockSize; ++i)
	{
		temp += c_block_A[i] * c_block_B[i];
	}

	result[tid] = temp;
}

__global__ void
array_vector_multiply_add(float *block_A, float *block_B, int blockSize, float *result, int n)
{
	const int tid = threadIdx.x + blockDim.x * blockIdx.x;

	if (tid >= n)
		return;

	const float *c_block_A = block_A + tid * blockSize;
	const float *c_block_B = block_B + tid * blockSize;

	float temp = 0;
	for (int i = 0; i < blockSize; ++i)
	{
		temp += c_block_A[i] * c_block_B[i];
	}

	result[tid] = temp;
}


__global__ void
vector_multiply_add(float *block_A, float *block_B, int blockSize, float *result, int n)
{
	const int tid = threadIdx.x + blockDim.x * blockIdx.x;

	if (tid >= n)
		return;

	const float *c_block_A = block_A + blockIdx.x * blockSize;
	const float *c_block_B = block_B + threadIdx.x * blockSize;

	float temp = 0;
	for (int i = 0; i < blockSize; ++i)
	{
		temp += c_block_A[i] * c_block_B[i];
	}

	result[tid] = temp;
}

__global__ void
vector_multiply_add(const float *blocks_A, const float *blocks_B, int block_B_groupSize, int blockSize, float *resultsBuffer)
{
	const int tid = threadIdx.x + blockDim.x * blockIdx.x;

	const int block_A_groupSize = 1;

	int blockGroupSize = block_A_groupSize * block_B_groupSize;

	int groupIndex = tid / blockGroupSize;

	int inGroupOffset = tid % blockGroupSize;

	int block_A_index = inGroupOffset / block_B_groupSize;

	int block_B_index = inGroupOffset % block_B_groupSize;

	const float *c_block_A = blocks_A + groupIndex * block_A_groupSize * blockSize + block_A_index * blockSize;
	const float *c_block_B = blocks_B + groupIndex * block_B_groupSize * blockSize + block_B_index * blockSize;

	float temp = 0;
	for (int i = 0; i < blockSize; ++i)
	{
		temp += c_block_A[i] * c_block_B[i];
	}

	resultsBuffer[tid] = temp;
}

__global__ void
vector_multiply_add(const float *blocks_A, const float *blocks_B, int block_B_groupSize, int blockSize, float *resultsBuffer, int n)
{
	const int tid = threadIdx.x + blockDim.x * blockIdx.x;

	if (tid >= n)
		return;

	const int block_A_groupSize = 1;

	int blockGroupSize = block_A_groupSize * block_B_groupSize;

	int groupIndex = tid / blockGroupSize;

	int inGroupOffset = tid % blockGroupSize;

	int block_A_index = inGroupOffset / block_B_groupSize;

	int block_B_index = inGroupOffset % block_B_groupSize;

	const float *c_block_A = blocks_A + groupIndex * block_A_groupSize * blockSize + block_A_index * blockSize;
	const float *c_block_B = blocks_B + groupIndex * block_B_groupSize * blockSize + block_B_index * blockSize;

	float temp = 0;
	for (int i = 0; i < blockSize; ++i)
	{
		temp += c_block_A[i] * c_block_B[i];
	}

	resultsBuffer[tid] = temp;
}

hipError_t standardize(float *sequence, int numberOfBlocks, int size, int numThreads, hipStream_t stream)
{
	standardize_block_kernel << <(numberOfBlocks + numThreads - 1) / numThreads, numThreads, 0, stream >> > (sequence, size, numberOfBlocks);
	hipError_t cuda_error = hipGetLastError();
	return cuda_error;
}

hipError_t standardize(float *sequence, int numberOfBlocks, int size, int numThreads)
{
	standardize_block_kernel << <(numberOfBlocks + numThreads - 1) / numThreads, numThreads>> > (sequence, size, numberOfBlocks);
	hipError_t cuda_error = hipGetLastError();
	return cuda_error;
}

hipError_t arrayMatchCc(float *A, float *B, float *C,
	int lengthOfArray, int numberOfArray,
	int numberOfProcessors, int numberOfThreads)
{
	hipError_t hipError_t = standardize(A, numberOfArray, lengthOfArray, numberOfThreads);

	if (hipError_t != hipSuccess)
		return hipError_t;

	hipError_t = standardize(B, numberOfArray, lengthOfArray, numberOfThreads);

	if (hipError_t != hipSuccess)
		return hipError_t;

	array_vector_multiply_add << <numberOfProcessors, numberOfThreads >> > (A, B, lengthOfArray, C, numberOfArray);
	hipError_t = hipGetLastError();

	return hipError_t;
}


hipError_t block_match_cc(float *blocks_A, float *blocks_B, int numBlocks_A, int numBlocks_B,
	int block_B_groupSize, int blockSize, float *result, int numProcessors, int numThreads, hipStream_t stream)
{
	hipError_t cuda_error = standardize(blocks_A, numBlocks_A, blockSize, numThreads, stream);

	if (cuda_error != hipSuccess)
		return cuda_error;

	cuda_error = standardize(blocks_B, numBlocks_B, blockSize, numThreads, stream);

	if (cuda_error != hipSuccess)
		return cuda_error;

	vector_multiply_add << <numProcessors, numThreads, 0, stream >> > (blocks_A, blocks_B, block_B_groupSize, blockSize, result);
	cuda_error = hipGetLastError();

	return cuda_error;
}

hipError_t block_match_cc_check_border(float *blocks_A, float *blocks_B, int numBlocks_A, int numBlocks_B,
	int block_B_groupSize, int blockSize, float *result, int numProcessors, int numThreads, hipStream_t stream)
{
	hipError_t cuda_error = standardize(blocks_A, numBlocks_A, blockSize, numThreads, stream);

	if (cuda_error != hipSuccess)
		return cuda_error;

	cuda_error = standardize(blocks_B, numBlocks_B, blockSize, numThreads, stream);

	if (cuda_error != hipSuccess)
		return cuda_error;

	vector_multiply_add << <numProcessors, numThreads, 0, stream >> > (blocks_A, blocks_B, block_B_groupSize, blockSize, result, numBlocks_B);
	cuda_error = hipGetLastError();

	return cuda_error;
}