#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

template <typename Type>
__global__ void
array_match_mse_kernel(const Type *block_A, const Type *block_B, int blockSize, Type *result)
{
	const int tid = threadIdx.x + blockDim.x * blockIdx.x;

	const Type *c_block_A = block_A + tid * blockSize;
	const Type *c_block_B = block_B + tid * blockSize;

	Type temp = 0;
	for (int i = 0;i<blockSize;++i)
	{
		temp += (c_block_A[i] - c_block_B[i]) * (c_block_A[i] - c_block_B[i]);
	}
	temp /= blockSize;
	result[tid] = temp;
}

template <typename Type>
__global__ void
array_match_mse_kernel(const Type *block_A, const Type *block_B, int blockSize, Type *result, int n)
{
	const int tid = threadIdx.x + blockDim.x * blockIdx.x;

	if (tid >= n)
		return;
	
	const Type *c_block_A = block_A + tid * blockSize;
	const Type *c_block_B = block_B + tid * blockSize;

	Type temp = 0;
	for (int i = 0; i<blockSize; ++i)
	{
		temp += (c_block_A[i] - c_block_B[i]) * (c_block_A[i] - c_block_B[i]);
	}
	temp /= blockSize;
	result[tid] = temp;
}

template <typename Type>
__global__ void
block_match_mse_async_kernel(const Type *blocks_A, const Type *blocks_B, int numberOfBlockBPerBlockA, int blockSize, Type *resultsBuffer)
{
	const int tid = threadIdx.x + blockDim.x * blockIdx.x;

	int groupIndex = tid / numberOfBlockBPerBlockA;

	int inGroupOffset = tid % numberOfBlockBPerBlockA;

	const Type *c_block_A = blocks_A + groupIndex * blockSize;
	const Type *c_block_B = blocks_B + groupIndex * numberOfBlockBPerBlockA * blockSize + inGroupOffset * blockSize;

	Type temp = 0;
	for (int i = 0; i<blockSize; ++i)
	{
		Type cc = c_block_A[i] - c_block_B[i];
		temp += cc*cc;
	}

	temp /= blockSize;

	resultsBuffer[tid] = temp;
}

template <typename Type>
__global__ void
block_match_mse_async_kernel(const Type *blocks_A, const Type *blocks_B, int numberOfBlockBPerBlockA, int blockSize, Type *resultsBuffer, int n)
{
	const int tid = threadIdx.x + blockDim.x * blockIdx.x;

	if (tid >= n)
		return;
		
	int groupIndex = tid / numberOfBlockBPerBlockA;

	int inGroupOffset = tid % numberOfBlockBPerBlockA;

	const Type *c_block_A = blocks_A + groupIndex * blockSize;
	const Type *c_block_B = blocks_B + groupIndex * numberOfBlockBPerBlockA * blockSize + inGroupOffset * blockSize;

	Type temp = 0;
	for (int i = 0; i<blockSize; ++i)
	{
		Type cc = c_block_A[i] - c_block_B[i];
		temp += cc*cc;
	}

	temp /= blockSize;

	resultsBuffer[tid] = temp;
}

template <typename Type>
hipError_t arrayMatchMse(Type *A, Type *B, Type *C,
	int lengthOfArray,
	int numberOfProcessors, int numberOfThreads)
{
	array_match_mse_kernel << <numberOfProcessors, numberOfThreads >> > (A, B, lengthOfArray, C);
	return hipGetLastError();
}

template <typename Type>
hipError_t arrayMatchMse(Type *A, Type *B, Type *C,
	int lengthOfArray, int numberOfArray,
	int numberOfProcessors, int numberOfThreads)
{
	array_match_mse_kernel << <numberOfProcessors, numberOfThreads >> > (A, B, lengthOfArray, C, numberOfArray);
	return hipGetLastError();
}

template <typename Type>
hipError_t block_match_mse(Type *blocks_A, Type *blocks_B, int numBlocks_A,
	int numberOfBlockBPerBlockA, int blockSize, Type *result, int numProcessors, int numThreads, hipStream_t stream)
{
	block_match_mse_async_kernel << <numProcessors, numThreads, 0, stream >> > (blocks_A, blocks_B, numberOfBlockBPerBlockA, blockSize, result);
	return hipGetLastError();
}

template <typename Type>
hipError_t block_match_mse_check_border(Type *blocks_A, Type *blocks_B, int numBlocks_A,
	int numberOfBlockBPerBlockA, int blockSize, Type *result, int numProcessors, int numThreads, hipStream_t stream)
{
	block_match_mse_async_kernel << <numProcessors, numThreads, 0, stream >> > (blocks_A, blocks_B, numberOfBlockBPerBlockA, blockSize, result, 
		numberOfBlockBPerBlockA * numBlocks_A);
	return hipGetLastError();
}

template
hipError_t block_match_mse(float *, float *, int,
	int, int, float *, int, int, hipStream_t);
template
hipError_t block_match_mse(double *, double *, int,
	int, int, double *, int, int, hipStream_t);
template
hipError_t block_match_mse_check_border(float *, float *, int,
	int, int, float *, int, int, hipStream_t);
template
hipError_t block_match_mse_check_border(double *, double *, int,
	int, int, double *, int, int, hipStream_t);
template
hipError_t arrayMatchMse(float *A, float *B, float *C,
	int lengthOfArray,
	int numberOfProcessors, int numberOfThreads);
template
hipError_t arrayMatchMse(double *A, double *B, double *C,
	int lengthOfArray,
	int numberOfProcessors, int numberOfThreads);
template
hipError_t arrayMatchMse(float *A, float *B, float *C,
	int lengthOfArray, int numberOfArray,
	int numberOfProcessors, int numberOfThreads);
template
hipError_t arrayMatchMse(double *A, double *B, double *C,
	int lengthOfArray, int numberOfArray,
	int numberOfProcessors, int numberOfThreads);